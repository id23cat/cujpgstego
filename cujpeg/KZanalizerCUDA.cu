#include "hip/hip_runtime.h"
/*
 * KZanalizerCUDA.cu
 *
 *  Created on: 02.05.2012
 *      Author: id23cat
 */

//#include "KZanalizerCUDA.cuh"
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "KZanalizer.h"
#include "Exceptions.h"


//#define CUDA_CALL(x) if ( x  != hipSuccess ) { \
		fprintf (stderr, " Error at %s :%d \n " , __FILE__ , __LINE__ ) ;\
		exit(EXIT_FAILURE) ;}

//KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 comp):
//KZanalizer(begin, end, comp){
////	colorComponent = comp;
////	// calculate elements count
////	INT16* ptr1 = begin.getCurBlock();
////	INT16* ptr2 = end.getCurBlock();
////
////	size_t count = (ptr2 - ptr1);
////	blockCount = count/64;
////
////	dctLen = blockCount * 8;
////	if(colorComponent != _ALL){
////		UINT8* decim = begin.getDecimation();
////		dctLen = dctLen * decim[colorComponent]/ (decim[_Y] + decim[_CB] + decim[_CR]);
////	}
////	SAFE_MALLOC_INT16(dctPtr, dctLen);
////
////
////	KZdataIterator kzit(dctPtr, dctLen, begin.getDecimation());
////	JPEG::DCTdataIterator it = begin;
////
//////	printf("KZ constructor\n");
////	while(it < end){
////		if(colorComponent == _ALL || colorComponent == it.color()){
////			for(int i=0; i<8; i++)
////				kzit[i] = it[KochZhaoZZ_order[i]];
//////			it.PrintBlock();
//////			printf("\n");
//////			kzit.PrintBlock();
//////			printf("\n");
////			kzit.mvToNextBlock();
////		}
////		it.mvToNextBlock();
////	}
//}

//__constant__ INT16 *dcts;

// CUDA SDK sum reduction:
// /home/id23cat/NVIDIA_GPU_Computing_SDK/C/src/reduction/doc/
//template <unsigned int blockSize>
//__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
//	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
//	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
//	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
//	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
//	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
//	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
//}
//#ifdef __DEVICE_EMULATION__
//#define EMUSYNC __syncthreads()
//#else
//#define EMUSYNC
//#endif

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
//template<class T>
//struct SharedMemory
//{
//    __device__ inline operator       T*()
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//
//    __device__ inline operator const T*() const
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//};

// specialize for double to avoid unaligned memory
// access compile errors
//template<>
//struct SharedMemory<double>
//{
//    __device__ inline operator       double*()
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//
//    __device__ inline operator const double*() const
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//};

/*
    This version is completely unrolled.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
//template <class T, unsigned int blockSize>
//__global__ void
//reduce5(T *g_idata, T *g_odata, unsigned int n)
//{
//    T *sdata = SharedMemory<T>();
//
//    // perform first level of reduction,
//    // reading from global memory, writing to shared memory
//    unsigned int tid = threadIdx.x;
//    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
//
//    T mySum = (i < n) ? g_idata[i] : 0;
//    if (i + blockSize < n)
//        mySum += g_idata[i+blockSize];
//
//    sdata[tid] = mySum;
//    __syncthreads();
//
//    // do reduction in shared mem
//    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
//    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
//    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
//
//#ifndef __DEVICE_EMULATION__
//    if (tid < 32)
//#endif
//    {
//        // now that we are using warp-synchronous programming (below)
//        // we need to declare our shared memory volatile so that the compiler
//        // doesn't reorder stores to it and induce incorrect behavior.
//        volatile T* smem = sdata;
//        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; EMUSYNC; }
//        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; EMUSYNC; }
//        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; EMUSYNC; }
//        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; EMUSYNC; }
//        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; EMUSYNC; }
//        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; EMUSYNC; }
//    }
//
//    // write result for this block to global mem
//    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
//}

#define PLUS(a, b) a += b
#define PLUSP(a, b) a += b*b
__global__ void GStd(INT16 *dct, VALUETYPE *pStd, VALUETYPE *pSum){
//	__shared__ VALUETYPE shsum[8];
//	__shared__ VALUETYPE shsum2[8];
//
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
//
//	INT16 val = dct[i];
//	VALUETYPE sum = val;
//	VALUETYPE sum2 = val;
//
//	val = dct[i+blockDim.x];
//	PLUS(sum, val);
//	PLUSP(sum2, val);
//
//	shsum[tid] = sum;
//	shsum2[tid] = sum2;
//	__syncthreads();
//
//	volatile VALUETYPE *smem = shsum;
//	volatile VALUETYPE *smem2 = shsum2;
//
//	PLUS(sum, smem[tid + 2]);
//	PLUSP(sum2, smem2[tid + 2]);
//
//	PLUS(sum, smem[tid + 1]);
//	PLUSP(sum2, smem2[tid + 1]);
	pSum[blockIdx.x] = 1;
	dct[threadIdx.x + blockDim.x*blockIdx.x] = threadIdx.x;
	if(tid == 0){
//		VALUETYPE mean = sum/8;
//		pSum[blockIdx.x] = sum;
//		pStd[blockIdx.x] = sqrtf(sum2/8 - mean*mean);
		pSum[blockIdx.x] = 1;
		pStd[blockIdx.x] = blockIdx.x;
	}
};

//__device__ VALUETYPE gProbability;

//__global__ void GMean(INT16 *dct, size_t dctlen, VALUETYPE *mean){
//
//};
//
//__global__ void GStd(INT16 *dct, size_t dctlen, VALUETYPE *std){
//
//};
//
//__global__ void GAnalize(INT16 *dct, size_t dctlen, VALUETYPE *probability){
//
//};
//

int KZanalizerCUDA::InitMem(){
	cutilSafeCall(
			hipMalloc(&dDCTptr, dctLen * sizeof(INT16)));
	cutilSafeCall(
			hipMemcpy(dDCTptr, dctPtr, dctLen * sizeof(INT16), hipMemcpyHostToDevice));
	cutilSafeCall(
			hipMalloc(&dMean, blockCount * sizeof(VALUETYPE)));
	cutilSafeCall(
			hipMalloc(&dStd, blockCount * sizeof(VALUETYPE)));
	cutilSafeCall(
			hipMalloc(&dSum, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&gHist, dctLen * sizeof(VALUETYPE)));


//	SAFE_MALLOC(hStd, blockCount, VALUETYPE);
	return dctLen;
}

bool KZanalizerCUDA::Analize(int Pthreshold ){
	InitMem();
	VALUETYPE *hStd = (VALUETYPE*) malloc( blockCount * sizeof(VALUETYPE));
	if(!hStd)
		fprintf(stderr, "hStd\n");
	VALUETYPE *hSum = (VALUETYPE*) malloc( blockCount * sizeof(VALUETYPE));
	if(!hSum)
		fprintf(stderr, "hSum\n");

	INT16 *m = (INT16*)malloc(dctLen * sizeof(INT16));

	for(int i=0; i<dctLen; i++)
			printf("DCT[%d]=%f\n", i, dctPtr[i]);


	dim3 blockSize(8);	//4
	dim3 gridSize(blockCount);
//	dim3 gridSize(10);
	GStd<<<gridSize, blockCount>>>(dDCTptr, dStd, dSum);

	cutilSafeCall(
				hipMemcpy(m, dDCTptr, dctLen * sizeof(INT16), hipMemcpyDeviceToHost));
	cutilSafeCall(
			hipMemcpy(hSum, dSum, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
	cutilSafeCall(
			hipMemcpy(hStd, dStd, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));

	for(int i=0; i<gridSize.x; i++)
		printf("Sum[%d]=%f, Std[%d]=%f\n", i, hSum[i], i, hStd[i]);
	for(int i=0; i<dctLen; i++)
			printf("M[%d]=%d\n", i, m[i]);

	printf("Bloks = %d", blockCount);
	free (hStd);
	free (hSum);
	return false;
}

KZanalizerCUDA::~KZanalizerCUDA(){
	cutilSafeCall(
			hipFree(dDCTptr));
	cutilSafeCall(
			hipFree(dMean));
	cutilSafeCall(
			hipFree(dStd));
	cutilSafeCall(
			hipFree(dSum));
//	SAFE_FREE(hSum);
//	SAFE_FREE(hStd);
}
