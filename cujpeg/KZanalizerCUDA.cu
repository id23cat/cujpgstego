#include "hip/hip_runtime.h"
/*
 * KZanalizerCUDA.cu
 *
 *  Created on: 02.05.2012
 *      Author: id23cat
 */

//#include "KZanalizerCUDA.cuh"
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "KZanalizer.h"

#include "cudefines.h"

//#define CUDA_CALL(x) if ( x  != hipSuccess ) { \
		fprintf (stderr, " Error at %s :%d \n " , __FILE__ , __LINE__ ) ;\
		exit(EXIT_FAILURE) ;}

//KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 comp):
//KZanalizer(begin, end, comp){
////	colorComponent = comp;
////	// calculate elements count
////	INT16* ptr1 = begin.getCurBlock();
////	INT16* ptr2 = end.getCurBlock();
////
////	size_t count = (ptr2 - ptr1);
////	blockCount = count/64;
////
////	dctLen = blockCount * 8;
////	if(colorComponent != _ALL){
////		UINT8* decim = begin.getDecimation();
////		dctLen = dctLen * decim[colorComponent]/ (decim[_Y] + decim[_CB] + decim[_CR]);
////	}
////	SAFE_MALLOC_INT16(dctPtr, dctLen);
////
////
////	KZdataIterator kzit(dctPtr, dctLen, begin.getDecimation());
////	JPEG::DCTdataIterator it = begin;
////
//////	printf("KZ constructor\n");
////	while(it < end){
////		if(colorComponent == _ALL || colorComponent == it.color()){
////			for(int i=0; i<8; i++)
////				kzit[i] = it[KochZhaoZZ_order[i]];
//////			it.PrintBlock();
//////			printf("\n");
//////			kzit.PrintBlock();
//////			printf("\n");
////			kzit.mvToNextBlock();
////		}
////		it.mvToNextBlock();
////	}
//}

//__constant__ INT16 *dcts;

// CUDA SDK sum reduction:
// /home/id23cat/NVIDIA_GPU_Computing_SDK/C/src/reduction/doc/
//template <unsigned int blockSize>
//__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
//	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
//	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
//	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
//	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
//	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
//	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
//}
//#ifdef __DEVICE_EMULATION__
//#define EMUSYNC __syncthreads()
//#else
//#define EMUSYNC
//#endif

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
//template<class T>
//struct SharedMemory
//{
//    __device__ inline operator       T*()
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//
//    __device__ inline operator const T*() const
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//};

// specialize for double to avoid unaligned memory
// access compile errors
//template<>
//struct SharedMemory<double>
//{
//    __device__ inline operator       double*()
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//
//    __device__ inline operator const double*() const
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//};

/*
    This version is completely unrolled.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
//template <class T, unsigned int blockSize>
//__global__ void
//reduce5(T *g_idata, T *g_odata, unsigned int n)
//{
//    T *sdata = SharedMemory<T>();
//
//    // perform first level of reduction,
//    // reading from global memory, writing to shared memory
//    unsigned int tid = threadIdx.x;
//    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
//
//    T mySum = (i < n) ? g_idata[i] : 0;
//    if (i + blockSize < n)
//        mySum += g_idata[i+blockSize];
//
//    sdata[tid] = mySum;
//    __syncthreads();
//
//    // do reduction in shared mem
//    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
//    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
//    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
//
//#ifndef __DEVICE_EMULATION__
//    if (tid < 32)
//#endif
//    {
//        // now that we are using warp-synchronous programming (below)
//        // we need to declare our shared memory volatile so that the compiler
//        // doesn't reorder stores to it and induce incorrect behavior.
//        volatile T* smem = sdata;
//        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; EMUSYNC; }
//        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; EMUSYNC; }
//        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; EMUSYNC; }
//        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; EMUSYNC; }
//        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; EMUSYNC; }
//        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; EMUSYNC; }
//    }
//
//    // write result for this block to global mem
//    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
//}

#define WARP_SIZE 32
#define HALF_WARP 16
#define HALF_KZBLOCK 4

#define PLUS(a, b) a += b
#define PLUS_SQ(a, b) a += b*b

__global__ void GStd(INT16 *dct, INT16 *psum=NULL, INT16 *psumsq=NULL,
		VALUETYPE *pStd=NULL, VALUETYPE *pSum=NULL){
	__shared__ INT16 shsum[4];
	__shared__ INT16 shsumsq[4];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tidx = threadIdx.x;
	unsigned int idxG = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	INT16 val = dct[idxG];
	INT16 sum = val;		// SUM
	INT16 sumsq = val;		// SUM OF SQUARES

	val = dct[idxG + blockDim.x];
	PLUS(sum, val);
	PLUS_SQ(sumsq, val);

	shsum[tidx] = sum;
	shsumsq[tidx] = sumsq;
	__syncthreads();

	volatile INT16 *smem = shsum;
	volatile INT16 *smemsq = shsumsq;

	if(tidx < 2){
		PLUS(smem[tidx], smem[tidx + 2]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 2]);
	}

	if(tidx == 0){
		PLUS(smem[tidx], smem[tidx + 1]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 1]);

		dct[idxG] = smem[tidx];


//		psum[blockIdx.x] = smem[0];
//		psumsq[blockIdx.x] = smemsq[tidx];
////		VALUETYPE mean = sum/8;
////		pSum[blockIdx.x] = sum;
////		pStd[blockIdx.x] = sqrtf(sum2/8 - mean*mean);
//		pSum[blockIdx.x] = 1;
//		pStd[blockIdx.x] = blockIdx.x;
	}
};

__global__ void GStd2(INT16 *dct, INT16 *psum=NULL, INT16 *psumsq=NULL,
		VALUETYPE *pStd=NULL, VALUETYPE *pSum=NULL){
	__shared__ INT16 shsum[512*2];
	__shared__ INT16 shsumsq[512*2];
//	__shared__ INT16* shsum;
//	__shared__ INT16* shsumsq;
////
////	The first thread in the block does the allocation
////	and then shares the pointer with all other threads
////	through shared memory, so that access can easily be coalesced.
//	if(threadIdx.x == 0){
//		shsum = (INT16*)malloc(blockDim.x * sizeof(INT16));
//		shsumsq = (INT16*)malloc(blockDim.x * sizeof(INT16));
//	}
//	__syncthreads();


	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tidx = threadIdx.x;
	unsigned int idxG = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	unsigned int BASEidx = tidx%WARP_SIZE >= HALF_WARP;
	unsigned int shidx = BASEidx + sizeof(INT16) * tidx;

	INT16 val = dct[idxG];
	INT16 sum = val;		// SUM
	INT16 sumsq = val;		// SUM OF SQUARES

	val = dct[idxG + HALF_KZBLOCK];
	PLUS(sum, val);
	PLUS_SQ(sumsq, val);

	shsum[shidx] = sum;
	shsumsq[shidx] = sumsq;
	__syncthreads();

	volatile INT16 *smem = shsum;
	volatile INT16 *smemsq = shsumsq;

	//// !!?????
	if(tidx < 2){
		PLUS(smem[tidx], smem[tidx + 2]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 2]);
	}

	if(tidx == 0){
		PLUS(smem[tidx], smem[tidx + 1]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 1]);

		dct[idxG] = smem[tidx];


//		psum[blockIdx.x] = smem[0];
//		psumsq[blockIdx.x] = smemsq[tidx];
////		VALUETYPE mean = sum/8;
////		pSum[blockIdx.x] = sum;
////		pStd[blockIdx.x] = sqrtf(sum2/8 - mean*mean);
//		pSum[blockIdx.x] = 1;
//		pStd[blockIdx.x] = blockIdx.x;
	}
};

//__global__ void DevTest(INT16 *dct){
//	int idx = blockIdx.x*(blockDim.x*2) +threadIdx.x;
//	dct[idx] = blockIdx.x;
//}


//template <typename FD>
//struct MEM{
//	FD *ptr;
//	size_t length;	// length in elements
//	MEM(): ptr(NULL), length(0){};
//	MEM(int VAL): ptr(NULL), length(VAL){};
//};

//typedef MEM<INT16> HOST_I16;
//typedef MEM<INT16> DEV_I16;
//typedef MEM<VALUETYPE> HOST_F32;
//typedef MEM<VALUETYPE> DEV_F32;

//#define MEM_H2D(H, D, TYPE) cutilSafeCall(hipMalloc(&D.ptr, dctLen * sizeof(INT16)));



//inline hipError_t HostToDev(DEV_I16 dst, HOST_I16 src = MEM<INT16>(0)){
//	if(dst.ptr == NULL){
//		if(dst.length > src.length)
//			cutilSafeCall(
//				hipMalloc(&dst.ptr, dst.length * sizeof(INT16)));
//		else if(src.length){
//			cutilSafeCall(
//				hipMalloc(&dst.ptr, src.length * sizeof(INT16)));
//			dst.length = src.length;
//		}
//		else return hipErrorOutOfMemory;
//	}else if(dst.length < src.length){
//		hipFree(dst.ptr);
//		cutilSafeCall(
//			hipMalloc(&dst.ptr, src.length * sizeof(INT16)));
//		dst.length = src.length;
//	}
//	if(src.ptr != NULL && src.length)
//		cutilSafeCall(
//			hipMemcpy(dst.ptr, src.ptr, dst.length * sizeof(INT16), hipMemcpyHostToDevice));
//
//	return hipSuccess;
//}

int KZanalizerCUDA::InitMem(){
	cutilSafeCall(
			hipMalloc(&dDCTptr, dctLen * sizeof(INT16)));
	cutilSafeCall(
			hipMemcpy(dDCTptr, dctPtr, dctLen * sizeof(INT16), hipMemcpyHostToDevice));
//	cutilSafeCall(
//			hipMalloc(&dMean, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&dStd, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&dSum, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&gHist, dctLen * sizeof(VALUETYPE)));


//	SAFE_MALLOC(hStd, blockCount, VALUETYPE);
	return dctLen;
}

bool KZanalizerCUDA::Analize(int Pthreshold ){
	TIMER_START();
	InitMem();

//	INT16 *dsum, *hsum, *dsumsq, *hsumsq;

//	SAFE_HOST_MALLOC(hsum, blockCount, INT16);
//	SAFE_HOST_MALLOC(hsumsq, blockCount, INT16);
//	SAFE_DEVICE_MALLOC(dsum, blockCount, INT16);
//	cutilSafeCall(hipMemset(dsum, 0, blockCount*sizeof(INT16)));
//	SAFE_DEVICE_MALLOC(dsumsq, blockCount, INT16);

	int shMpT = 2*sizeof(INT16);	// shared memory per thread in bytes;
	int thcount = ColcMaxThreadsPerBLock(shMpT, 8, dctLen * sizeof(INT16), 4);
	int blkcount = CalcBlockCount(shMpT, dctLen * sizeof(INT16), thcount);
	printf("Threads count = %d, blocks count = %d (totMem = %d)\n",
			thcount, blkcount, dctLen * sizeof(INT16));


	dim3 blockSize(4);	//4
	dim3 gridSize(blockCount);
////	dim3 gridSize(10);
//	GStd<<<gridSize, blockCount>>>(dDCTptr, dStd, dSum);
//	GStd<<<gridSize, blockSize>>>(dDCTptr, dsum, dsumsq);
	GStd<<<gridSize, blockSize>>>( dDCTptr );

	TIMER_STOP("GPU STD");

	INT16 *ppp;
	SAFE_HOST_MALLOC(ppp, dctLen, INT16);
//	COPY_TO_HOST(hsum, dsum, blockCount, INT16);
//	COPY_TO_HOST(hsumsq, dsumsq, blockCount, INT16);
	COPY_TO_HOST(ppp, dDCTptr, dctLen, INT16);

//	for(int i=0,k=0,j=0; i<dctLen; i++){
//		printf("DCT[%d]=%d DCT[%d]=%d\n", i, dctPtr[i], i, ppp[i]);
//		k++;
//		if( k== 8){
////			printf("\t SUM[%d]=%d, SUMSQ[%d]=%d\n", j, hsum[j], j, hsumsq[j]);
//			printf("\t[%d]=%d\n", j, ppp[i-7]);
//			j++;
//			k=0;
//		}
//	}


//
//	cutilSafeCall(
//				hipMemcpy(m, dDCTptr, dctLen * sizeof(INT16), hipMemcpyDeviceToHost));
//	cutilSafeCall(
//			hipMemcpy(hSum, dSum, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
//	cutilSafeCall(
//			hipMemcpy(hStd, dStd, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
//
//	for(int i=0; i<gridSize.x; i++)
//		printf("Sum[%d]=%f, Std[%d]=%f\n", i, hSum[i], i, hStd[i]);
//	for(int i=0; i<dctLen; i++)
//			printf("M[%d]=%d\n", i, m[i]);
//
//	printf("Bloks = %d", blockCount);
//	free (hStd);
//	free (hSum);
	return false;
}

KZanalizerCUDA::~KZanalizerCUDA(){
	SAFE_DEVICE_FREE(dDCTptr);
//	cutilSafeCall(
//			hipFree(dMean));
//	cutilSafeCall(
//			hipFree(dStd));
//	cutilSafeCall(
//			hipFree(dSum));
//	SAFE_HOST_FREE(hSum);
//	SAFE_HOST_FREE(hStd);
}
