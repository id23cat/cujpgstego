#include "hip/hip_runtime.h"
/*
 * KZanalizerCUDA.cu
 *
 *  Created on: 02.05.2012
 *      Author: id23cat
 */

//#include "KZanalizerCUDA.cuh"
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "KZanalizer.h"
#include "datatypes.h"
#include "cudefines.h"
#ifdef TIME_COMPARE
#include "Timer.h"
#endif

#if (__CUDA_ARCH__ < 200)
#define THREADS 256
#else
#define THREADS 512
#endif
#define SHMEM THREADS*8

//__align__(128) INT16 *dDCTptr;			// pointer in device memory

//#define CUDA_CALL(x) if ( x  != hipSuccess ) { \
		fprintf (stderr, " Error at %s :%d \n " , __FILE__ , __LINE__ ) ;\
		exit(EXIT_FAILURE) ;}

//KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 comp):
//KZanalizer(begin, end, comp){
////	colorComponent = comp;
////	// calculate elements count
////	INT16* ptr1 = begin.getCurBlock();
////	INT16* ptr2 = end.getCurBlock();
////
////	size_t count = (ptr2 - ptr1);
////	blockCount = count/64;
////
////	dctLen = blockCount * 8;
////	if(colorComponent != _ALL){
////		UINT8* decim = begin.getDecimation();
////		dctLen = dctLen * decim[colorComponent]/ (decim[_Y] + decim[_CB] + decim[_CR]);
////	}
////	SAFE_MALLOC_INT16(dctPtr, dctLen);
////
////
////	KZdataIterator kzit(dctPtr, dctLen, begin.getDecimation());
////	JPEG::DCTdataIterator it = begin;
////
//////	printf("KZ constructor\n");
////	while(it < end){
////		if(colorComponent == _ALL || colorComponent == it.color()){
////			for(int i=0; i<8; i++)
////				kzit[i] = it[KochZhaoZZ_order[i]];
//////			it.PrintBlock();
//////			printf("\n");
//////			kzit.PrintBlock();
//////			printf("\n");
////			kzit.mvToNextBlock();
////		}
////		it.mvToNextBlock();
////	}
//}

//__constant__ INT16 *dcts;

// CUDA SDK sum reduction:
// /home/id23cat/NVIDIA_GPU_Computing_SDK/C/src/reduction/doc/
//template <unsigned int blockSize>
//__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
//	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
//	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
//	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
//	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
//	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
//	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
//}
//#ifdef __DEVICE_EMULATION__
//#define EMUSYNC __syncthreads()
//#else
//#define EMUSYNC
//#endif

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
//template<class T>
//struct SharedMemory
//{
//    __device__ inline operator       T*()
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//
//    __device__ inline operator const T*() const
//    {
//        extern __shared__ int __smem[];
//        return (T*)__smem;
//    }
//};

// specialize for double to avoid unaligned memory
// access compile errors
//template<>
//struct SharedMemory<double>
//{
//    __device__ inline operator       double*()
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//
//    __device__ inline operator const double*() const
//    {
//        extern __shared__ double __smem_d[];
//        return (double*)__smem_d;
//    }
//};

/*
    This version is completely unrolled.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
//template <class T, unsigned int blockSize>
//__global__ void
//reduce5(T *g_idata, T *g_odata, unsigned int n)
//{
//    T *sdata = SharedMemory<T>();
//
//    // perform first level of reduction,
//    // reading from global memory, writing to shared memory
//    unsigned int tid = threadIdx.x;
//    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
//
//    T mySum = (i < n) ? g_idata[i] : 0;
//    if (i + blockSize < n)
//        mySum += g_idata[i+blockSize];
//
//    sdata[tid] = mySum;
//    __syncthreads();
//
//    // do reduction in shared mem
//    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
//    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
//    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
//
//#ifndef __DEVICE_EMULATION__
//    if (tid < 32)
//#endif
//    {
//        // now that we are using warp-synchronous programming (below)
//        // we need to declare our shared memory volatile so that the compiler
//        // doesn't reorder stores to it and induce incorrect behavior.
//        volatile T* smem = sdata;
//        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; EMUSYNC; }
//        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; EMUSYNC; }
//        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; EMUSYNC; }
//        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; EMUSYNC; }
//        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; EMUSYNC; }
//        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; EMUSYNC; }
//    }
//
//    // write result for this block to global mem
//    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
//}

#define WARP_SIZE 32
#define HALF_WARP 16
#define HALF_KZBLOCK 4

#define BANK_COUNT 16	// 32

#define PLUS(a, b) a += b
#define PLUS_SQ(a, b) a += b*b

/******************* GStd1 *********************/
__global__ void GStd(INT16 *dct, INT16 *psum=NULL, INT16 *psumsq=NULL,
		VALUETYPE *pStd=NULL, VALUETYPE *pSum=NULL){
	__shared__ INT16 shsum[4];
	__shared__ INT16 shsumsq[4];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tidx = threadIdx.x;
	unsigned int idxG = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	INT16 val = dct[idxG];
	INT16 sum = val;		// SUM
	INT16 sumsq = val;		// SUM OF SQUARES

	val = dct[idxG + blockDim.x];
	PLUS(sum, val);
	PLUS_SQ(sumsq, val);

	shsum[tidx] = sum;
	shsumsq[tidx] = sumsq;
	__syncthreads();

	volatile INT16 *smem = shsum;
	volatile INT16 *smemsq = shsumsq;

	if(tidx < 2){
		PLUS(smem[tidx], smem[tidx + 2]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 2]);
	}

	if(tidx == 0){
		PLUS(smem[tidx], smem[tidx + 1]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 1]);

		dct[idxG] = smem[tidx];


//		psum[blockIdx.x] = smem[0];
//		psumsq[blockIdx.x] = smemsq[tidx];
////		VALUETYPE mean = sum/8;
////		pSum[blockIdx.x] = sum;
////		pStd[blockIdx.x] = sqrtf(sum2/8 - mean*mean);
//		pSum[blockIdx.x] = 1;
//		pStd[blockIdx.x] = blockIdx.x;
	}
};


/******************* GStd2 *********************/
__global__ void GStd2(INT16 *dct/*, INT16 *psum=NULL, INT16 *psumsq=NULL,
		VALUETYPE *pStd=NULL, VALUETYPE *pSum=NULL*/){
	__shared__ INT16 shsum[512*2];
	__shared__ INT16 shsumsq[512*2];
//	__shared__ INT16* shsum;
//	__shared__ INT16* shsumsq;
////
////	The first thread in the block does the allocation
////	and then shares the pointer with all other threads
////	through shared memory, so that access can easily be coalesced.
//	if(threadIdx.x == 0){
//		shsum = (INT16*)malloc(blockDim.x * sizeof(INT16));
//		shsumsq = (INT16*)malloc(blockDim.x * sizeof(INT16));
//	}
//	__syncthreads();


	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tidx = threadIdx.x;
	unsigned int idxG = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	// if tidx
//	unsigned int BASEidx = tidx%WARP_SIZE >= HALF_WARP;
//	unsigned int shidx = BASEidx + sizeof(INT16) * tidx;

	// for 1.x arch
	unsigned int warpBlock = WARP_SIZE;			// 1.x elements count in the warp
//	unsigned int warpSize = 2*WARP_SIZE;		// 2.x
//	unsigned int BASEidx1x = tidx % WARP_SIZE >= BANK_COUNT;		// BANK_COUNT=16
//	unsigned int BASEidx2x = tidx % (2*WARP_SIZE) >= BANK_COUNT;	// BANK_COUNT=32

	unsigned int BASEidx1x = tidx % warpBlock >= BANK_COUNT;
	BASEidx1x += (unsigned int)tidx/warpBlock * warpBlock;

	// !!!
	unsigned int shidx = BASEidx1x + sizeof(INT16) * (tidx%1);

	INT16 val = dct[idxG];
	INT16 sum = val;		// SUM
	INT16 sumsq = val;		// SUM OF SQUARES

	val = dct[idxG + HALF_KZBLOCK];
	PLUS(sum, val);
	PLUS_SQ(sumsq, val);

	shsum[shidx] = sum;
	shsumsq[shidx] = sumsq;
	__syncthreads();

	volatile INT16 *smem = shsum;
	volatile INT16 *smemsq = shsumsq;

	//// !!?????
	if(tidx < 2){
		PLUS(smem[tidx], smem[tidx + 2]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 2]);
	}

	if(tidx == 0){
		PLUS(smem[tidx], smem[tidx + 1]);
		PLUS_SQ(smemsq[tidx], smemsq[tidx + 1]);

		dct[idxG] = smem[tidx];


//		psum[blockIdx.x] = smem[0];
//		psumsq[blockIdx.x] = smemsq[tidx];
////		VALUETYPE mean = sum/8;
////		pSum[blockIdx.x] = sum;
////		pStd[blockIdx.x] = sqrtf(sum2/8 - mean*mean);
//		pSum[blockIdx.x] = 1;
//		pStd[blockIdx.x] = blockIdx.x;
	}
};


/******************* GStd3 *********************/
typedef struct{
	INT32 x;
	INT32 y;
	INT32 z;
	INT32 w;
} my_uint4;

typedef struct sh2{
	INT16 x;
	INT16 y;
	__device__ inline sh2(UINT32 i){x = i&0xFFFF0000; y = i&0x0000FFFF;};
//	__device__ inline UINT32 toint(){return (UINT32)x<<16+y;};

} my_short2;

typedef struct{
	INT16 x0;
	INT16 x1;
	INT16 y0;
	INT16 y1;
	INT16 z0;
	INT16 z1;
	INT16 w0;
	INT16 w1;
} my_int4;

#define ALIGN_UP(offset, alignment) \
(offset) = ((offset) + (alignment) – 1) & ~((alignment) – 1)

__global__ void GStd3(INT16 *dct){
	my_int4 *ptr = (my_int4*)dct;
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	my_int4 ui = ptr[tidx];


	ui.x0 += ui.x1;
	ui.x0 += ui.y0;
	ui.x0 += ui.y1;
	ui.x0 += ui.z0;
	ui.x0 += ui.z1;
	ui.x0 += ui.w0;
	ui.x0 += ui.w1;
	__syncthreads();

	ptr[tidx] = ui;
}

#define FIRST(X) X >> 16
#define LAST(X) X & 0x0000FFFF

__global__ void GStd4(INT16 *dct){
//	uint4 *ptr = (uint4*)dct;

	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	uint4 ui4 = ((uint4*)dct)[tidx];

	INT16 v = 0;

	v += FIRST(ui4.x);
	v += LAST(ui4.x);

	v += FIRST(ui4.y);
	v += LAST(ui4.y);

	v += FIRST(ui4.z);
	v += LAST(ui4.z);

	v += FIRST(ui4.w);
	v += LAST(ui4.w);

	ui4.x = (UINT32)v;

//	__syncthreads();

	((uint4*)dct)[tidx] = ui4;
}

__global__ void GStd5(INT16 *dct){

	__shared__ INT16 shmem[SHMEM];	//256*8 //4096// 8*512
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	((uint4*)shmem)[threadIdx.x] = ((uint4*)dct)[tidx];

	int shidx = threadIdx.x*8;
	INT16 v = 0;

//#pragma unroll 4
//	for(int i=0; i<8; i++)
//		v += shmem[shidx+i];
	v += shmem[shidx];
	v += shmem[shidx+1];

	v += shmem[shidx+2];
	v += shmem[shidx+3];

	v += shmem[shidx+4];
	v += shmem[shidx+5];

	v += shmem[shidx+6];
	v += shmem[shidx+7];

	shmem[shidx+0] = v;

	((uint4*)dct)[tidx] = ((uint4*)shmem)[threadIdx.x];
}

__device__ inline void SumSum(INT16 val, INT16 &sum, INT32 &sumsq){
	sum +=val;
	sumsq += val*val;
}

__global__ void GStd5_2(INT16 *dct){

	__shared__ INT16 shmem[SHMEM];	//256*8 //4096// 8*512
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	((uint4*)shmem)[threadIdx.x] = ((uint4*)dct)[tidx];

	int shidx = threadIdx.x << 3;		// threadIdx.x*8;
	INT16 sum = 0;
	INT32 sumsq = 0;

//#pragma unroll 4
//	for(int i=0; i<8; i++)
//		v += shmem[shidx+i];

	SumSum( shmem[shidx], sum, sumsq);
	SumSum( shmem[shidx+1], sum, sumsq);

	SumSum(shmem[shidx+2], sum, sumsq);
	SumSum(shmem[shidx+3], sum, sumsq);

	SumSum(shmem[shidx+4], sum, sumsq);
	SumSum(shmem[shidx+5], sum, sumsq);

	SumSum(shmem[shidx+6], sum, sumsq);
	SumSum(shmem[shidx+7], sum, sumsq);

	shmem[shidx+0] = sum;
	shmem[shidx+1] = sumsq;

	((uint4*)dct)[tidx] = ((uint4*)shmem)[threadIdx.x];
}

__global__ void GStd6(INT16 *dct){

	__shared__ INT16 shmem[SHMEM];	//256*8 //4096// 8*512
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	((uint4*)shmem)[threadIdx.x] = ((uint4*)dct)[tidx];

//	int shidx = threadIdx.x*8;
	int shidx = threadIdx.x << 3;
	INT16 v = 0;
//	int i0 = threadIdx.x%16 >> 2 << 1;		// ((( thid % 16 ) / 4 ) * 2 )
	int i0 = ( threadIdx.x - threadIdx.x >> 4) >> 2 << 1;

//	int idx = 0;
//#pragma unroll
//	for(int i=0; i<8; i++){
////		idx = ( i0 + i ) % 8;
//		int k = i0 + i;
//		idx = k -( k >> 3);
//		v += shmem[ shidx + idx ];
//	}

	int idx = i0 - (i0 >> 3);
	v += shmem[shidx + idx];
	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];

	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];
	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];

	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];
	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];

	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];
	i0++; idx = i0 - (i0 >> 3);
	v += shmem[shidx+idx];

	shmem[shidx] = v;

	((uint4*)dct)[tidx] = ((uint4*)shmem)[threadIdx.x];
}

//typedef MEM<INT16> HOST_I16;
//typedef MEM<INT16> DEV_I16;
//typedef MEM<VALUETYPE> HOST_F32;
//typedef MEM<VALUETYPE> DEV_F32;

//#define MEM_H2D(H, D, TYPE) cutilSafeCall(hipMalloc(&D.ptr, dctLen * sizeof(INT16)));

//inline hipError_t HostToDev(DEV_I16 dst, HOST_I16 src = MEM<INT16>(0)){
//	if(dst.ptr == NULL){
//		if(dst.length > src.length)
//			cutilSafeCall(
//				hipMalloc(&dst.ptr, dst.length * sizeof(INT16)));
//		else if(src.length){
//			cutilSafeCall(
//				hipMalloc(&dst.ptr, src.length * sizeof(INT16)));
//			dst.length = src.length;
//		}
//		else return hipErrorOutOfMemory;
//	}else if(dst.length < src.length){
//		hipFree(dst.ptr);
//		cutilSafeCall(
//			hipMalloc(&dst.ptr, src.length * sizeof(INT16)));
//		dst.length = src.length;
//	}
//	if(src.ptr != NULL && src.length)
//		cutilSafeCall(
//			hipMemcpy(dst.ptr, src.ptr, dst.length * sizeof(INT16), hipMemcpyHostToDevice));
//
//	return hipSuccess;
//}

int KZanalizerCUDA::InitMem(){
	cutilSafeCall(
			hipMalloc(&dDCTptr, dctLen * sizeof(INT16)));
	cutilSafeCall(
			hipMemcpy(dDCTptr, dctPtr, dctLen * sizeof(INT16), hipMemcpyHostToDevice));
//	cutilSafeCall(
//			hipMalloc(&dMean, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&dStd, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&dSum, blockCount * sizeof(VALUETYPE)));
//	cutilSafeCall(
//			hipMalloc(&gHist, dctLen * sizeof(VALUETYPE)));


//	SAFE_MALLOC(hStd, blockCount, VALUETYPE);
	return dctLen;
}

bool KZanalizerCUDA::Analize(int Pthreshold ){

	InitMem();

//	int shMpT = 2*sizeof(INT16);	// shared memory per thread in bytes;
//	int thcount = ColcMaxThreadsPerBLock(shMpT, 8, dctLen * sizeof(INT16), 4);
//	int blkcount = CalcBlockCount(shMpT, dctLen * sizeof(INT16), thcount);
//	printf("Threads count = %d, blocks count = %d (totMem = %d)\n",
//			thcount, blkcount, dctLen * sizeof(INT16));


//	dim3 blockSize(4);	//4
//	dim3 gridSize(blockCount);
////	dim3 gridSize(10);
//	GStd<<<gridSize, blockCount>>>(dDCTptr, dStd, dSum);
//	GStd<<<gridSize, blockSize>>>(dDCTptr, dsum, dsumsq);


//	GStd<<<gridSize, blockSize>>>( dDCTptr );

	int threads = THREADS;
	printf("Threads count = %d, blocks count = %d\n", threads, blockCount/threads+1);
#ifdef TIME_COMPARE
	Timer timer;
	timer.Start();
#endif
	int iMAX=1000;
	for(int i=0; i<iMAX; i++){
//	GStd3<<<blockCount/threads+1, threads>>>( dDCTptr );
//	GStd3<<<4, 4>>>( dDCTptr );
//	GStd4<<<blockCount/threads+1, threads>>>( dDCTptr );
//	GStd5<<<blockCount/threads+1, threads>>>( dDCTptr );

	GStd5_2<<<blockCount/threads+1, threads>>>( dDCTptr );
//	GStd6<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	float time1 = timer.Stop()/iMAX;
	printf("GPU STD5_2: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd6<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD6: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd2<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD2: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd3<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD3: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd4<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD4: %.5fs\n", time1);
#endif

#ifdef TIME_COMPARE
	timer.Start();
#endif
	for(int i=0; i<iMAX; i++){
		GStd5<<<blockCount/threads+1, threads>>>( dDCTptr );
	}
#ifdef TIME_COMPARE
	time1 = timer.Stop()/iMAX;
	printf("GPU STD5: %.5fs\n", time1);
#endif

	INT16 *ppp;
	SAFE_HOST_MALLOC(ppp, dctLen, INT16);
//	COPY_TO_HOST(hsum, dsum, blockCount, INT16);
//	COPY_TO_HOST(hsumsq, dsumsq, blockCount, INT16);
	COPY_TO_HOST(ppp, dDCTptr, dctLen, INT16);

//	for(int i=0,k=0,j=0; i<dctLen; i++){
//		printf("DCT[%d]=%d DCT[%d]=%d\n", i, dctPtr[i], i, ppp[i]);
//		k++;
//		if( k== 8){
////			printf("\t SUM[%d]=%d, SUMSQ[%d]=%d\n", j, hsum[j], j, hsumsq[j]);
//			printf("\t[%d]=%d\n", j, ppp[i-7]);
//			j++;
//			k=0;
//		}
//	}


//
//	cutilSafeCall(
//				hipMemcpy(m, dDCTptr, dctLen * sizeof(INT16), hipMemcpyDeviceToHost));
//	cutilSafeCall(
//			hipMemcpy(hSum, dSum, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
//	cutilSafeCall(
//			hipMemcpy(hStd, dStd, blockCount * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
//
//	for(int i=0; i<gridSize.x; i++)
//		printf("Sum[%d]=%f, Std[%d]=%f\n", i, hSum[i], i, hStd[i]);
//	for(int i=0; i<dctLen; i++)
//			printf("M[%d]=%d\n", i, m[i]);
//
//	printf("Bloks = %d", blockCount);
//	free (hStd);
//	free (hSum);
	return false;
}

KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 component):
KZanalizer(begin, end, component){
	hipDeviceReset();
};

KZanalizerCUDA::~KZanalizerCUDA(){
	SAFE_DEVICE_FREE(dDCTptr);
//	cutilSafeCall(
//			hipFree(dMean));
//	cutilSafeCall(
//			hipFree(dStd));
//	cutilSafeCall(
//			hipFree(dSum));
//	SAFE_HOST_FREE(hSum);
//	SAFE_HOST_FREE(hStd);
}
