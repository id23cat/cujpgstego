#include "hip/hip_runtime.h"
/*
 * KZanalizerCUDA.cu
 *
 *  Created on: 02.05.2012
 *      Author: id23cat
 */

//#include "KZanalizerCUDA.cuh"
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "KZanalizer.h"
#include "datatypes.h"
#include "cudefines.h"
#ifdef TIME_COMPARE
#include "Timer.h"
#endif

#if (__CUDA_ARCH__ < 200)
#define THREADS 256
#else
#define THREADS 512
#endif
#define SHMEM THREADS*8



#define WARP_SIZE 32
#define HALF_WARP 16
#define HALF_KZBLOCK 4

#define BANK_COUNT 16	// 32

#define PLUS(a, b) a += b
#define PLUS_SQ(a, b) a += b*b



__global__ void GStd(INT16 *dct){

	__shared__ INT16 shmem[SHMEM];	//256*8 //4096// 8*512
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	((uint4*)shmem)[threadIdx.x] = ((uint4*)dct)[tidx];

	int shidx = threadIdx.x*8;
	INT16 v = 0;

//#pragma unroll 4
//	for(int i=0; i<8; i++)
//		v += shmem[shidx+i];
	v += shmem[shidx];
	v += shmem[shidx+1];

	v += shmem[shidx+2];
	v += shmem[shidx+3];

	v += shmem[shidx+4];
	v += shmem[shidx+5];

	v += shmem[shidx+6];
	v += shmem[shidx+7];

	shmem[shidx+0] = v;

	((uint4*)dct)[tidx] = ((uint4*)shmem)[threadIdx.x];
}

__device__ inline void SumSum(INT16 val, INT16 &sum, INT32 &sumsq){
	sum +=val;
	sumsq += val*val;
}



int KZanalizerCUDA::InitMem(){
	cutilSafeCall(
			hipMalloc(&dDCTptr, dctLen * sizeof(INT16)));
	cutilSafeCall(
			hipMemcpy(dDCTptr, dctPtr, dctLen * sizeof(INT16), hipMemcpyHostToDevice));

	return dctLen;
}

bool KZanalizerCUDA::Analize(int Pthreshold ){

	InitMem();

	int threads = THREADS;
	printf("Threads count = %d, blocks count = %d\n", threads, blockCount/threads+1);

#ifdef TIME_COMPARE
	Timer timer;
	timer.Start();
#endif

		GStd<<<blockCount/threads+1, threads>>>( dDCTptr );

#ifdef TIME_COMPARE
	float time = timer.Stop();
	printf("GPU STD5: %.5fs\n", time);
#endif

	INT16 *ppp;
	SAFE_HOST_MALLOC(ppp, dctLen, INT16);

	COPY_TO_HOST(ppp, dDCTptr, dctLen, INT16);

	return false;
}

KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 component):
KZanalizer(begin, end, component){
	hipDeviceReset();
};

KZanalizerCUDA::~KZanalizerCUDA(){
	SAFE_DEVICE_FREE(dDCTptr);
}
