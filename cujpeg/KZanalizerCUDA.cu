#include "hip/hip_runtime.h"
/*
 * KZanalizerCUDA.cu
 *
 *  Created on: 02.05.2012
 *      Author: id23cat
 */

//#include "KZanalizerCUDA.cuh"
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "KZanalizer.h"
#include "datatypes.h"
#include "cudefines.h"
#ifdef TIME_COMPARE
#include "Timer.h"
#endif

#if (__CUDA_ARCH__ < 200)
#define THREADS 256
#else
#define THREADS 512
#endif
#define SHMEM THREADS*8



#define WARP_SIZE 32
#define HALF_WARP 16
#define HALF_KZBLOCK 4

#define BANK_COUNT 16	// 32

#define PLUS(a, b) a += b
#define PLUS_SQ(a, b) a += b*b

#define HISTSIZE 1024
__shared__ float shvalue[HISTSIZE];
__shared__ int shcount[HISTSIZE];
__shared__ int shactual_sz;

class GPUhist{
	float *value;
	int *count;
	int *actual_size;
public:
	__device__ GPUhist();
	__device__ void AddValue(float val);
private:
	__device__ void Exchange(int i1, int i2);

};

__device__ GPUhist::GPUhist():
				value(shvalue),
				count(shcount),
				actual_size(&shactual_sz){

	int dim = blockDim.x;
	int idx = threadIdx.x;

	int elems_per_thread = HISTSIZE/dim;

	for(int i=0, j=0; j<elems_per_thread; j++, i += dim){
		value[idx + i] = 0;
		count[idx + i] = 0;
	}

	if(idx==0) actual_size = 0;
};

__device__ void GPUhist::AddValue(float v){
	int i=0;
	for(; i<*actual_size; i++){
		if (value[i] == v) {
			atomicAdd(&(count[i]), 1);
			if (i > 0 && count[i - 1] < count[i])
				Exchange(i - 1, i);
			return;
		}
	}
	if(i < HISTSIZE){
		atomicAdd(&(value[i]), v);
		atomicAdd(&(count[i]), 1);
	}
}

__device__ void GPUhist::Exchange(int i1, int i2){
	count[i1] = atomicExch(&(count[i2]), count[i1]);
	value[i1] = atomicExch(&(value[i2]), value[i1]);
}


__global__ void GStd(INT16 *dct){

	__shared__ INT16 shmem[SHMEM];	//256*8 //4096// 8*512
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;
	((uint4*)shmem)[threadIdx.x] = ((uint4*)dct)[tidx];

	int shidx = threadIdx.x*8;
	INT16 v = 0;

//#pragma unroll 4
//	for(int i=0; i<8; i++)
//		v += shmem[shidx+i];
	v += shmem[shidx];
	v += shmem[shidx+1];

	v += shmem[shidx+2];
	v += shmem[shidx+3];

	v += shmem[shidx+4];
	v += shmem[shidx+5];

	v += shmem[shidx+6];
	v += shmem[shidx+7];

	shmem[shidx+0] = v;

	((uint4*)dct)[tidx] = ((uint4*)shmem)[threadIdx.x];
}

__device__ inline void SumSum(INT16 val, INT16 &sum, INT32 &sumsq){
	sum +=val;
	sumsq += val*val;
}



int KZanalizerCUDA::InitMem(){
	cutilSafeCall(
			hipMalloc(&dDCTptr, dctLen * sizeof(INT16)));
	cutilSafeCall(
			hipMemcpy(dDCTptr, dctPtr, dctLen * sizeof(INT16), hipMemcpyHostToDevice));

	return dctLen;
}

bool KZanalizerCUDA::Analize(int Pthreshold ){

	InitMem();

	int threads = THREADS;
	printf("Threads count = %d, blocks count = %d\n", threads, blockCount/threads+1);

#ifdef TIME_COMPARE
	Timer timer;
	timer.Start();
#endif

		GStd<<<blockCount/threads+1, threads>>>( dDCTptr );

#ifdef TIME_COMPARE
	float time = timer.Stop();
	printf("GPU STD5: %.5fs\n", time);
#endif

	INT16 *ppp;
	SAFE_HOST_MALLOC(ppp, dctLen, INT16);

	COPY_TO_HOST(ppp, dDCTptr, dctLen, INT16);

	return false;
}

KZanalizerCUDA::KZanalizerCUDA(JPEG::DCTdataIterator begin, JPEG::DCTdataIterator end, UINT8 component):
KZanalizer(begin, end, component){
	hipDeviceReset();
};

KZanalizerCUDA::~KZanalizerCUDA(){
	SAFE_DEVICE_FREE(dDCTptr);
}
